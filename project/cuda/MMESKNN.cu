#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>
#include <stdlib.h>
#include <iostream>
#include <chrono>
#include "MMESKNN.hpp"
using namespace std;


#include <opencv2/video/background_segm.hpp>
#include <opencv2/core/core.hpp>
#include <opencv2/imgproc/imgproc.hpp>
#include <opencv2/highgui/highgui.hpp>
using namespace cv;

//{ to do - paralelization ...
//struct KNNInvoker....
__device__ void _cvUpdatePixelBackgroundNP(
    const uchar *currPixel,
    int channels,
    int nSample,
    bool  *flag,
    uchar *Model,
    uchar *NextLongUpdate,
    uchar *NextMidUpdate,
    uchar *NextShortUpdate,
    uchar *ModelIndexLong,
    uchar *ModelIndexMid,
    uchar *ModelIndexShort,
    int LongCounter,
    int MidCounter,
    int ShortCounter,
    int LongUpdate,
    int MidUpdate,
    int ShortUpdate,
    bool include,
    unsigned int seed,
    hiprandState_t *states
)
{
    // hold the offset
    long flagoffsetShort = *ModelIndexShort;
    long flagoffsetMid   = *ModelIndexMid  + nSample * 1;
    long flagoffsetLong  = *ModelIndexLong + nSample * 2;
    long offsetShort = channels * ( *ModelIndexShort );
    long offsetMid   = channels * ( *ModelIndexMid  + nSample * 1 );
    long offsetLong  = channels * ( *ModelIndexLong + nSample * 2 );
    // uint seed = time( NULL );

    /* we have to initialize the state */
    hiprand_init( seed, /* the seed controls the sequence of random values that are produced */
                 blockIdx.x, /* the sequence number is only important with multiple cores */
                 0, /* the offset is how much extra we advance in the sequence for each call, can be 0 */
                 &states[blockIdx.x] );

    // Long update? --------------------------------------------------------------------------------------
    if ( *NextLongUpdate == LongCounter )
    {
        // add the oldest pixel from Mid to the list of values (for each color)
        Model[offsetLong]     = Model[offsetMid];
        Model[offsetLong + 1] = Model[offsetMid + 1];
        Model[offsetLong + 2] = Model[offsetMid + 2];
        flag[flagoffsetLong]  = flag[flagoffsetMid];
        // increase the index
        *ModelIndexLong = ( *ModelIndexLong >= ( nSample - 1 ) ) ? 0 : ( *ModelIndexLong + 1 );
    }
    if ( LongCounter == ( LongUpdate - 1 ) )
    {
        *NextLongUpdate = ( uchar )( hiprand( &states[blockIdx.x] ) % LongUpdate ); //0,...LongUpdate-1;
    }

    // Mid update? --------------------------------------------------------------------------------------
    if ( *NextMidUpdate == MidCounter )
    {
        // add this pixel to the list of values (for each color)
        Model[offsetMid]     = Model[offsetShort];
        Model[offsetMid + 1] = Model[offsetShort + 1];
        Model[offsetMid + 2] = Model[offsetShort + 2];
        flag[flagoffsetMid]  = flag[flagoffsetShort];
        // increase the index
        *ModelIndexMid = ( *ModelIndexMid >= ( nSample - 1 ) ) ? 0 : ( *ModelIndexMid + 1 );
    }
    if ( MidCounter == ( MidUpdate - 1 ) )
    {
        *NextMidUpdate = ( uchar )( hiprand( &states[blockIdx.x] ) % MidUpdate );
    }

    // Short update? --------------------------------------------------------------------------------------
    if ( *NextShortUpdate == ShortCounter )
    {
        // add this pixel to the list of values (for each color)
        Model[offsetShort]     = currPixel[0];
        Model[offsetShort + 1] = currPixel[1];
        Model[offsetShort + 2] = currPixel[2];
        flag[flagoffsetShort]  = include;
        // increase the index
        *ModelIndexShort = ( *ModelIndexShort >= ( nSample - 1 ) ) ? 0 : ( *ModelIndexShort + 1 );
    }
    if ( ShortCounter == ( ShortUpdate - 1 ) )
    {
        *NextShortUpdate = ( uchar )( hiprand( &states[blockIdx.x] ) % ShortUpdate );
    }
}

__device__ int _cvCheckPixelBackgroundNP(
    const uchar *currPixel,
    int channels,
    int nSample,
    bool *flag,
    uchar *Model,
    float Tb,
    int kNN,
    float tau,
    bool ShadowDetection,
    bool *include
)
{
    int Pbf = 0; // the total probability that this pixel is background
    int Pb = 0; //background model probability

    *include = false; //do we include this pixel into background model?

    /* long posPixel = pixel * ndata * nSample * 3; */
    // now increase the probability for each pixel
    for ( int n = 0; n < nSample * 3; n++ )
    {
        //calculate difference and distance
        int d0 = Model[n * channels] - currPixel[0];
        int d1 = Model[n * channels + 1] - currPixel[1];
        int d2 = Model[n * channels + 2] - currPixel[2];
        int dist2 = d0 * d0 + d1 * d1 + d2 * d2;

        if ( dist2 < Tb )
        {
            Pbf++;//all
            //background only
            if ( flag[n] ) //indicator
            {
                Pb++;
                if ( Pb >= kNN ) //Tb
                {
                    *include = true; //include
                    return 1;//background ->exit , dst[pixel]=0
                }
            }
        }
    }

    //include?
    if ( Pbf >= kNN ) //Tbf)
    {
        *include = true;
    }

    // Detected as moving object, perform shadow detection
    if ( ShadowDetection )
    {
        int Ps = 0; // the total probability that this pixel is background shadow
        for ( int n = 0; n < nSample * 3; n++ )
        {
            uchar *mean_m = Model + n * channels;

            if ( flag[n] ) //check only background
            {
                float numerator = 0.0f;
                float denominator = 0.0f;
                for ( int c = 0; c < channels; c++ )
                {
                    numerator   += currPixel[c] * mean_m[c];
                    denominator += mean_m[c] * mean_m[c];
                }

                // no division by zero allowed
                if ( denominator == 0 )
                {
                    return 0; //dst[pixel]=255
                }

                // if tau < a < 1 then also check the color distortion
                if ( numerator <= denominator && numerator >= tau * denominator )
                {
                    float a = numerator / denominator;
                    float dist2a = 0.0f;

                    for ( int c = 0; c < channels; c++ )
                    {
                        float dD = a * mean_m[c] - currPixel[c];
                        dist2a += dD * dD;
                    }

                    if ( dist2a < Tb * a * a )
                    {
                        Ps++;
                        if ( Ps >= kNN ) //shadow
                        {
                            return 2; //dst[[pixel]=ShadowValue
                        }
                    }
                }
            }
        }
    }
    return 0; //dst[pixel]=255
}

__global__ void icvUpdatePixelBackgroundNP(
    int cols,
    int rows,
    int channels,
    int totalPixels,
    uchar *srcData,
    uchar *dst,
    bool *flag,
    uchar *Model,
    uchar *NextLongUpdate,
    uchar *NextMidUpdate,
    uchar *NextShortUpdate,
    uchar *ModelIndexLong,
    uchar *ModelIndexMid,
    uchar *ModelIndexShort,
    int LongCounter,
    int MidCounter,
    int ShortCounter,
    int LongUpdate,
    int MidUpdate,
    int ShortUpdate,
    int nSample,
    float Tb,
    int kNN,
    float Tau,
    bool ShadowDetection,
    uchar ShadowValue,
    unsigned int seed,
    hiprandState_t *states
)
{
    /* 2D */
    int posCol = blockIdx.x * blockDim.x + threadIdx.x;
    int posRow = blockIdx.y * blockDim.y + threadIdx.y;
    int posPixel = cols * ( posRow - 1 ) + posCol;
    /* 1D */
    /* int posPixel = blockIdx.x * blockDim.x + threadIdx.x; */
    uchar *currPixel = srcData + posPixel * channels;
    //GPU parallel
    if ( posPixel < totalPixels && posCol < cols && posRow < rows )
    {
        // int posPixel = ncols * y + x;
        /* start addr of current pixel */

        //update model+ background subtract
        bool include = 0;
        int result = _cvCheckPixelBackgroundNP(
                         currPixel,
                         channels,
                         nSample,
                         flag + posPixel * nSample * 3,
                         Model + posPixel * channels * nSample * 3,
                         // pass Model's start address of pixel
                         Tb,
                         kNN,
                         Tau,
                         ShadowDetection,
                         &include
                     );

        _cvUpdatePixelBackgroundNP(
            currPixel,
            channels,
            nSample,
            flag + posPixel * nSample * 3,
            Model + posPixel * channels * nSample * 3,
            NextLongUpdate + posPixel,
            NextMidUpdate + posPixel,
            NextShortUpdate + posPixel,
            ModelIndexLong + posPixel,
            ModelIndexMid + posPixel,
            ModelIndexShort + posPixel,
            LongCounter,
            MidCounter,
            ShortCounter,
            LongUpdate,
            MidUpdate,
            ShortUpdate,
            include,
            seed,
            states
        );
        switch ( result )
        {
        case 0:
            //foreground
            dst[posPixel] = 255;
            break;
        case 1:
            //background
            dst[posPixel] = 0;
            break;
        case 2:
            //shadow
            dst[posPixel] = ShadowValue;
            break;
        }
    }

}



void MMESKNN::apply( cv::Mat &image, cv::Mat &dst, double learningRate )
{
    bool needToInitialize = nframes == 0 || learningRate >= 1 || image.size() != frameSize || image.type() != frameType;
    if ( needToInitialize )
    {
        initialize( image.size(), image.type() );
    }

    dst.create( image.size(), CV_8UC1 );

    ++nframes;
    learningRate = learningRate >= 0 && nframes > 1 ? learningRate : 1. / std::min( 2 * nframes, history );
    CV_Assert( learningRate >= 0 );

    // recalculate update rates - in case alpha is changed
    // calculate update parameters (using alpha)
    int Kshort, Kmid, Klong;
    //approximate exponential learning curve
    Kshort = ( int )( log( 0.7 ) / log( 1 - learningRate ) ) + 1; //Kshort
    Kmid   = ( int )( log( 0.4 ) / log( 1 - learningRate ) ) - Kshort + 1; //Kmid
    Klong  = ( int )( log( 0.1 ) / log( 1 - learningRate ) ) - Kshort - Kmid + 1; //Klong

    //refresh rates
    int ShortUpdate = ( Kshort / nSample ) + 1;
    int MidUpdate   = ( Kmid   / nSample ) + 1;
    int LongUpdate  = ( Klong  / nSample ) + 1;

    // cuda
    //rows:size().height ; cols:size().width
    int totalPixels = image.rows * image.cols;
    hipMemcpy( d_imageData, image.ptr(), sizeof( uchar ) * totalPixels * image.channels(), hipMemcpyHostToDevice );
    //hipMemcpy(d_dstData,   dst.ptr(),   sizeof(uchar) * totalPixels, hipMemcpyHostToDevice);

    /* icvUpdatePixelBackgroundNP <<< ( totalPixels + 255 ) / 256, 256 >>> ( */
    /* icvUpdatePixelBackgroundNP <<< ( totalPixels + 1023) / 1024, 1024 >>> ( */
    /* dim3 threadsPerBlock( 32, 32 ); */
    /* our video resolution 16:9 */
    dim3 threadsPerBlock( 32, 18 );
    dim3 numBlocks( image.cols + threadsPerBlock.x - 1 / threadsPerBlock.x, image.rows + threadsPerBlock.y - 1 / threadsPerBlock.y );
    icvUpdatePixelBackgroundNP <<<numBlocks, threadsPerBlock>>> (
        image.cols,
        image.rows,
        image.channels(),
        totalPixels,
        d_imageData,
        d_dstData,
        d_flag,
        d_bgmodel,
        d_nNextLongUpdate,
        d_nNextMidUpdate,
        d_nNextShortUpdate,
        d_aModelIndexLong,
        d_aModelIndexMid,
        d_aModelIndexShort,
        nLongCounter,
        nMidCounter,
        nShortCounter,
        LongUpdate,
        MidUpdate,
        ShortUpdate,
        nSample,
        fTb,
        nkNN,
        fTau,
        ShadowDetection, // 1: do ShadowDetection
        ShadowValue, // default = (uchar) 127
        time( NULL ),
        states
    );

    // hipMemcpy(image.ptr(), d_imageData, sizeof(uchar) * totalPixels * image.channels(), hipMemcpyDeviceToHost);
    hipMemcpy( dst.ptr(),   d_dstData,   sizeof( uchar ) * totalPixels, hipMemcpyDeviceToHost );
    // hipMemcpy(bgmodel, d_bgmodel, sizeof(uchar) * totalPixels * image.channels() * nSample * 3, hipMemcpyDeviceToHost);
    // hipMemcpy(aModelIndexShort, d_aModelIndexShort, sizeof(uchar) * totalPixels, hipMemcpyDeviceToHost);
    // hipMemcpy(aModelIndexMid  , d_aModelIndexMid  , sizeof(uchar) * totalPixels, hipMemcpyDeviceToHost);
    // hipMemcpy(aModelIndexLong , d_aModelIndexLong , sizeof(uchar) * totalPixels, hipMemcpyDeviceToHost);
    // hipMemcpy(nNextShortUpdate, d_nNextShortUpdate, sizeof(uchar) * totalPixels, hipMemcpyDeviceToHost);
    // hipMemcpy(nNextMidUpdate  , d_nNextMidUpdate  , sizeof(uchar) * totalPixels, hipMemcpyDeviceToHost);
    // hipMemcpy(nNextLongUpdate , d_nNextLongUpdate , sizeof(uchar) * totalPixels, hipMemcpyDeviceToHost);
    // hipMemcpy(flag, d_flag, sizeof(bool) * nSample * 3 * totalPixels, hipMemcpyDeviceToHost);

    //int i;
    //for(i=0; i<totalPixels; i++){
    //    printf("%d\n", dst.data[i]);
    //}
    //printf("--------------------------------------------------\n");

    //update counters for the refresh rate
    //0,1,...,ShortUpdate-1
    if ( ++nShortCounter >= ShortUpdate )
    {
        nShortCounter = 0;
    }
    if ( ++nMidCounter >= MidUpdate )
    {
        nMidCounter = 0;
    }
    if ( ++nLongCounter >= LongUpdate )
    {
        nLongCounter = 0;
    }
}

int main( int argc, char *argv[] )
{

    Mat frame;
    Mat output;

    auto start = std::chrono::system_clock::now();
    MMESKNN *BG = new MMESKNN();
    /* Ptr<BackgroundSubtractor> BG = createBackgroundSubtractorMOG2(); */
    /* Ptr<BackgroundSubtractor> BG = createBackgroundSubtractorKNN(); */

    VideoCapture input( argv[1] );
    if ( argc == 3 )
    {
        input.set( CV_CAP_PROP_POS_FRAMES, atoi( argv[2] ) * 30 );
    }
    std::chrono::duration<double> BGtime = std::chrono::duration<double>::zero();
    while ( true )
    {
        if ( !( input.read( frame ) ) ) //get one frame form video
        {
            break;
        }
        auto t1 = std::chrono::system_clock::now();
        BG->apply( frame, output );
        auto t2 = std::chrono::system_clock::now();
        BGtime += t2 - t1;
        imshow( "Origin", frame );
        imshow( "KNN",    output );
        if ( waitKey( 30 ) >= 0 )
        {
            break;
        }
    }
    std::chrono::duration<double> totalTime = std::chrono::system_clock::now() - start;
    cout << "BG time: " << BGtime.count() << "s\n";
    cout << "total time: " << totalTime.count() << "s\n";
    delete BG;
}
